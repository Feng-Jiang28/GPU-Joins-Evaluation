#include "hip/hip_runtime.h"
#define HIPCUB_STDERR

#include <iostream>
#include <vector>
#include <tuple>
#include <chrono>
#include <unistd.h>
#include <fstream>

#include <hip/hip_runtime.h>

#include "tuple.cuh"
#include "utils.cuh"
#include "operators.cuh"
#include "tpc_utils.hpp"
#include "../data_gen/generator.cuh"
#include "sort_merge_join.cuh"
#include "partitioned_hash_join.cuh"
#include "sort_hash_join.cuh"
#include "experiment_util.cuh"
#include "join_base.hpp"

using namespace std;

/**
The code is designed to run a specific query (referred to as "q64") using different join algorithms on two data tables.
It loads the data, performs the join operation using the specified algorithm, and then outputs the results and performance statistics.

Main Steps
Load the Data: Load columns from binary files.
Initialize and Open Scans: Set up scan operators to read the data.
Perform the Join: Use the specified join algorithm to combine the tables.
Output Results: Print the result and performance statistics.
Profile and Cleanup: Save the performance statistics to a file and free memory.
*/

// Define types for the columns
#define PAYLOAD_COL_T long
#define KEY_COL_T long

// Macro to load a column from a binary file
#define LOAD_Q64_COL(dir, p, N, from, to, seed) \
    to* p; \
    GET_DATA_TYPE(p) \
    { \
        std::string filename = dir + "/" + #p + ".bin"; \
        read_col<from,to>(filename, p, (N), true, (seed)); \
    }

// Macro to load all required columns
#define LOAD_Q64_COL_ALL(dir, Nss, Ncd) \
    LOAD_Q64_COL((dir), ss_cdemo_sk, (Nss), int, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_customer_sk, (Nss), long, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_addr_sk, (Nss), long, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_item_sk, (Nss), long, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_ticket_number, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_wholesale_cost, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_list_price, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), ss_coupon_amt, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), d_year, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), s_store_name, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), s_zip, (Nss), long, PAYLOAD_COL_T, 42) \
    LOAD_Q64_COL((dir), cd_demo_sk, (Ncd), int, KEY_COL_T, 42) \
    LOAD_Q64_COL((dir), cd_marital_status, (Ncd), long, PAYLOAD_COL_T, 42)

// Function to run the q64 query with a specified algorithm
void run_q64(std::string algo, std::string profile_output_) {
    const int Nss = 57898426;
    const int Ncd = 1920800;
    std::string dir(TPC_DATA_PREFIX"tpcds_sf100/q64");

    // Load all columns
    LOAD_Q64_COL_ALL(dir, Nss, Ncd)

    // Define the chunk types for the tables and the join result
    using ss_t = struct Chunk<ss_cdemo_sk_t,ss_customer_sk_t,ss_addr_sk_t,ss_item_sk_t,ss_ticket_number_t,ss_wholesale_cost_t,ss_list_price_t,ss_coupon_amt_t,d_year_t,s_store_name_t,s_zip_t>;
    using cd_t = struct Chunk<cd_demo_sk_t, cd_marital_status_t>;
    
    using join_t = struct Chunk<cd_demo_sk_t,cd_marital_status_t,ss_customer_sk_t,ss_addr_sk_t,ss_item_sk_t,ss_ticket_number_t,ss_wholesale_cost_t,ss_list_price_t,ss_coupon_amt_t,d_year_t,s_store_name_t,s_zip_t>;

    // Create tuples from the columns
    auto cd_cols = std::make_tuple(cd_demo_sk, cd_marital_status);
    auto ss_cols = std::make_tuple(ss_cdemo_sk,ss_customer_sk,ss_addr_sk,ss_item_sk,ss_ticket_number,ss_wholesale_cost,ss_list_price,ss_coupon_amt,d_year,s_store_name,s_zip);

    // Initialize scan operators to read the data
    ScanOperator<cd_t> cd_scan(std::move(cd_cols), Ncd, Ncd);
    ScanOperator<ss_t> ss_scan(std::move(ss_cols), Nss, Nss);

    // Open scans and read data
    cd_scan.open(); ss_scan.open();
    auto relation_cd = cd_scan.next();
    auto relation_ss = ss_scan.next();
    cd_scan.close(); ss_scan.close();

    const int circular_buffer_size = Nss;  // Size of the circular buffer
    const int first_bit = 0;
    const int log_part1 = 9;
    const int log_part2 = 6;

    // Pointer to the join implementation
    JoinBase<join_t> *impl;
    // Select the join algorithm based on the input argument
    if (algo == "SMJ") {
        impl = new SortMergeJoin<cd_t, ss_t, join_t, true>(relation_cd, relation_ss, circular_buffer_size);
    } else if (algo == "PHJ") {
        impl = new PartitionHashJoin<cd_t, ss_t, join_t>(relation_cd, relation_ss, log_part1, log_part2, first_bit, circular_buffer_size);
    } else if (algo == "SHJ") {
        impl = new SortHashJoin<cd_t, ss_t, join_t>(relation_cd, relation_ss, first_bit, log_part1+log_part2, circular_buffer_size);
    } else if (algo == "SMJI") {
        impl = new SortMergeJoinByIndex<cd_t, ss_t, join_t>(relation_cd, relation_ss, circular_buffer_size);
    } 
    else {
        std::cout << "Invalid algorithm name: " << algo << std::endl;
        exit(1);
    }

    // Perform the join operation
    auto out = impl->join();
    hipDeviceSynchronize();

    cout << "\nOutput Cardinality = " << out.num_items << endl;
    cout << "Results (first 10 items): \n";
    out.peek(min(10, out.num_items));

    // Print performance statistics
    impl->print_stats();

    // Save performance statistics to a file
    std::ofstream fout;
    fout.open(profile_output_, ios::app);
    fout << get_utc_time() << ","
        << relation_cd.num_items << "," << relation_ss.num_items << ","
        << algo << ",";

    auto stats = impl->all_stats();
    for(auto t : stats) {
        fout << t << ",";
    }

    fout << std::endl;
    fout.close();

    // Free memory
    relation_cd.free_mem();
    relation_ss.free_mem();
    out.free_mem();
}

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cout << "Usage: ./bin/volcano/q64 <algo> <profile_output>" << std::endl;
        exit(1);
    }
    std::string algo = argv[1]; // Join algorithm to use
    std::string profile_output = argv[2]; // Output file for performance statistics
    run_q64(algo, profile_output); // Run the query
    return 0;
}