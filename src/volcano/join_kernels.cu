// In join_kernels.cu
#include "join_kernels.cuh"

template<tyname T>
void sort_on_gpu(T* keys, int num_items){
    T* d_keys;
    T* d_sorted_keys;
    hipMalloc(&d_keys, sizeof(T)*num_items);
    hipMemcpy(d_keys, keys, sizeof(T)*num_items, hipMemcpyDefault);
    hipMalloc(&d_sorted_keys, sizeof(T)*num_items);

    void* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortKeys(nullptr, temp_storage_bytes, d_keys, d_sorted_keys, num_items);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys, d_sorted_keys, num_items);

    hipMemcpy(keys, d_sorted_keys, sizeof(T)*num_items, hipMemcpyDefault);
    hipFree(d_keys);
    hipFree(d_sorted_keys);
    hipDeviceSynchronize();
}

// Explicit instantiation for types you'll use
template void sort_on_gpu<int>(int* keys, int num_items);
template void sort_on_gpu<long>(long *keys, int num_items);
// Add other types as needed.