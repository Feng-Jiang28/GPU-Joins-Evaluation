#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <tuple>
#include <chrono>
#include <unistd.h>
#include <fstream>

#include "../volcano/operators.cuh"
#include "experiment_util.cuh"


#include "cudf_join.hpp"
using namespace std;

enum Input {
    RelR,
    RelS,
    UniqueKeys
};

std::string get_path_name(enum Input table, const struct join_args& args) {
    auto nr = args.nr;
    auto ns = args.ns;
    auto uk = args.unique_keys;

#ifndef KEY_T_8B
    std::string subfolder = "int/";
#else
    std::string subfolder = "long/";
#endif

    if(table == UniqueKeys) {
        return args.data_path_prefix+subfolder+"r_" + std::to_string(uk) + ".bin";
    }

    if(args.type == PK_FK) {
        return table == RelR ? args.data_path_prefix+subfolder+"r_" + std::to_string(nr) + ".bin"
                             : args.data_path_prefix+subfolder+"s_" + std::to_string(nr) + "_" +std::to_string(ns) + "_" + (args.dist == UNIFORM ? "uniform" : "zipf_") + (args.dist == UNIFORM ? "" : std::to_string(args.zipf_factor))+".bin";
    }
    else {
        return table == RelR ? args.data_path_prefix+subfolder+"s_" + std::to_string(args.unique_keys) + "_" +std::to_string(nr) + "_uniform.bin"
                             : args.data_path_prefix+subfolder+"s_" + std::to_string(args.unique_keys) + "_" +std::to_string(ns) + "_uniform.bin";
    }
}

template<typename join_key_t, typename col_t, typename TupleR, typename TupleS>
void prepare_workload(const struct join_args& args, TupleR& relation_r, TupleS& relation_s) {
    constexpr int R_NUM_COLS = TupleR::num_cols, S_NUM_COLS = TupleS::num_cols;

    auto nr = args.nr;
    auto ns = args.ns;

    join_key_t *rkeys = nullptr, *skeys = nullptr;
    std::array<col_t*, R_NUM_COLS-1> r;
    std::array<col_t*, S_NUM_COLS-1> s;

    std::string rpath = get_path_name(RelR, args);
    std::string spath = get_path_name(RelS, args);

    if(args.type == PK_FK) {
        // create relation R
        if(input_exists(rpath)) {
            cout << "R read from disk\n";
            alloc_load_column(rpath, rkeys, nr);
        } else {
            create_integral_relation_unique(&rkeys, nr, false, static_cast<join_key_t>(0), true, 42);
            write_to_disk(rkeys, nr, rpath);
        }

        // create relation S
        if(input_exists(spath)) {
            cout << "S read from disk\n";
            alloc_load_column(spath, skeys, ns);
        } else {
            if(args.dist == UNIFORM) {
                create_fk_from_pk_uniform(&skeys, ns, rkeys, nr);
            }
            else {
                create_fk_from_pk_zipf(&skeys, ns, rkeys, nr, args.zipf_factor);
            }

            write_to_disk(skeys, ns, spath);
        }
    }
    else if(args.type == FK_FK) {
        if(args.dist == ZIPF) {
            std::cout << "FKFK join with zipf distribution is not supported for now\n";
            std::exit(-1);
        }

        join_key_t* uk = nullptr;
        auto nuk = args.unique_keys;
        if(!input_exists(rpath) || !input_exists(spath)) {
            std::string upath = get_path_name(UniqueKeys, args);
            if(input_exists(upath)) {
                cout << "Unique keys read from disk\n";
                alloc_load_column(upath, uk, nuk);
            } else {
                create_integral_relation_unique(&uk, nuk, false, static_cast<join_key_t>(0), true, 42);
                write_to_disk(uk, nuk, upath);
            }
        }

        if(input_exists(rpath)) {
            cout << "R read from disk\n";
            alloc_load_column(rpath, rkeys, nr);
        } else {
            create_fk_from_pk_uniform(&rkeys, nr, uk, nuk);
            write_to_disk(rkeys, nr, rpath);
        }

        // create relation S
        if(input_exists(spath)) {
            cout << "S read from disk\n";
            alloc_load_column(spath, skeys, ns);
        } else {
            create_fk_from_pk_uniform(&skeys, ns, uk, nuk);
            write_to_disk(skeys, ns, spath);
        }
    }
    else {
        std::cout << "Unsupported join type\n";
        std::exit(-1);
    }

#ifdef MR_FILTER_FK
    if(args.selectivity > 1) {
        if(args.selectivity >= args.nr) assert(false);
        #pragma omp parallel for
        for(int i = 0; i < nr; i++) {
            if(i % args.selectivity == 0) continue;
            rkeys[i] += (1<<30);
        }
    }
    std::cout << "Filtered FK to reduce the match ratio\n";
#endif

#ifdef SORTED_REL
    sort_on_gpu(rkeys, nr);
    sort_on_gpu(skeys, ns);
#endif

    if(sizeof(col_t) == sizeof(join_key_t)) {
        for(int i = 0; i < S_NUM_COLS-1; i++) {
            s[i] = new col_t[ns];
            memcpy(s[i], skeys, sizeof(col_t)*ns);
        }

        for(int i = 0; i < R_NUM_COLS-1; i++) {
            r[i] = new col_t[nr];
            memcpy(r[i], rkeys, sizeof(col_t)*nr);
        }
    } else {
        for(int i = 0; i < S_NUM_COLS-1; i++) {
            s[i] = new col_t[ns];
        }

        for(int i = 0; i < R_NUM_COLS-1; i++) {
            r[i] = new col_t[nr];
        }

        #pragma unroll
        for(int i = 0; i < ns; i++) {
            s[0][i] = static_cast<col_t>(skeys[i]);
        }

        #pragma unroll
        for(int i = 0; i < nr; i++) {
            r[0][i] = static_cast<col_t>(rkeys[i]);
        }

        for(int i = 1; i < S_NUM_COLS-1; i++) {
            memcpy(s[i], s[0], sizeof(col_t)*ns);
        }

        for(int i = 1; i < R_NUM_COLS-1; i++) {
            memcpy(r[i], r[0], sizeof(col_t)*nr);
        }
    }

    cout << "Data preparation is done\n";

    auto b_cols = std::tuple_cat(std::make_tuple(rkeys), std::tuple_cat(r));
    auto p_cols = std::tuple_cat(std::make_tuple(skeys), std::tuple_cat(s));

    ScanOperator<TupleR> op1(std::move(b_cols), nr, nr);
    ScanOperator<TupleS> op2(std::move(p_cols), ns, ns);

    op1.open(); op2.open();
    relation_r = op1.next();
    relation_s = op2.next();
    op1.close(); op2.close();

    // adjust the match ratio
    // if the match ratio is 1 out of M,
    // then we randomly remove floor(|R|/M) elements from relation R (assuming M < |R|)
    // this is simulating the filtering before join
#ifndef MR_FILTER_FK
    if(args.selectivity > 1) {
        if(args.selectivity >= args.nr) assert(false);
        relation_r.num_items /= args.selectivity;
    }
    cout << "The effective |R| after adjusting the selectivity is " << relation_r.num_items << endl;
#endif

    release_mem(relation_r.select_vec);
    release_mem(relation_s.select_vec);
    relation_r.select_vec = nullptr;
    relation_s.select_vec = nullptr;

    delete[] rkeys;
    delete[] skeys;
    for(int i = 0; i < R_NUM_COLS-1; i++) {
        delete [] r[i];
    }

    for(int i = 0; i < S_NUM_COLS-1; i++) {
        delete [] s[i];
    }
}


void print_usage() {
    cout << "Join Microbenchmarks\n";
    cout << "Usage: <binary> [-l|-h] -r <log_2(|R|)> -s <log_2(|S|)> -m <R payload cols> -n <S payload cols> -t <join type> -d <distribution> -z <zipf factor> -o <output file> -f <data path prefix> -e <selectivity> -u <unique keys> -i <join algorithm> -p <phj log part1> -q <phj log part2>\n";
    cout << "Options:\n";
    cout << "-l: use log scale for |R|, |S|, and unique keys. Default: no.\n";
    cout << "-h: print this message\n";
    cout << "-r: log_2(|R|) if using -l flag otherwise the actual size\n";
    cout << "-s: log_2(|S|) if using -l flag otherwise the actual size\n";
    cout << "-m: number of payload columns in R\n";
    cout << "-n: number of payload columns in S\n";
    cout << "-t: join type, pkfk or fkfk. Default: pkfk\n";
    cout << "-d: distribution type, uniform or zipf. Default: uniform\n";
    cout << "-z: zipf factor, only valid when -d zipf is used\n";
    cout << "-o: output file name\n";
    cout << "-f: path to the generated data directory if any; otherwise provide a location where you want the generated data to be stored\n";
    cout << "-e: selectivity, only valid when -t pkfk is used. Default: 1.\n";
    cout << "-u: number of unique keys, only valid when -t fkfk is used\n";
    cout << "-i: join algorithm, phj, shj, smj, smji (case sensitive)\n";
    cout << "-p: log_2(partitions in 1st pass) for PHJ. Default: 9.\n";
    cout << "-q: log_2(partitions in 2nd pass) for PHJ. Default: 6.\n";
    cout << "(Note: -p and -q are only valid when -i phj or -i shj is used)\n";
    cout << "Example: ./bin/volcano/join_exp -l -r 12 -s 12 -m 1 -n 1 -t pkfk -d uniform -o join_exp.csv -f /home/data/ -e 1 -i phj -p 9 -q 6\n";
}

void parse_args(int argc, char** argv, struct join_args& args) {
    bool use_log_scale = false;
    for(;;)
    {
      switch(getopt(argc, argv, "r:s:v:m:n:t:d:z:o:e:u:i:p:q:f:alh"))
      {
        case 'r':
            args.nr = atoi(optarg);
            continue;
        case 's':
            args.ns = atoi(optarg);
            continue;
        case 'v':
            args.vec_size = atoi(optarg);
            continue;
        case 'm':
            args.pr = atoi(optarg);
            continue;
        case 'n':
            args.ps = atoi(optarg);
            continue;
        case 't':
            if(strcasecmp(optarg, "fkfk") == 0) {
                args.type = FK_FK;
            }
            continue;
        case 'd':
            if(strcasecmp(optarg, "zipf") == 0) {
                args.dist = ZIPF;
            }
            else {
                args.zipf_factor = 0.0f;
            }
            continue;
        case 'z':
            args.zipf_factor = atof(optarg);
            continue;
        case 'o':
            args.output = std::string(optarg);
            continue;
        case 'f':
            args.data_path_prefix = std::string(optarg);
            if(args.data_path_prefix.back() != '/') args.data_path_prefix += "/";
            continue;
        case 'e':
            args.selectivity = atoi(optarg);
            continue;
        case 'u':
            args.unique_keys = atoi(optarg);
            continue;
        case 'a':
            args.agg_only = true;
            continue;
        case 'l':
            use_log_scale = true;
            continue;
        case 'i':
            if(std::string(optarg) == "phj") args.algo = PHJ;
            else if(std::string(optarg) == "shj") args.algo = SHJ;
            else if(std::string(optarg) == "smji") args.algo = SMJI;
            else args.algo = SMJ;
            continue;
        case 'p':
            args.phj_log_part1 = atoi(optarg);
            continue;
        case 'q':
            args.phj_log_part2 = atoi(optarg);
            continue;
        case 'h':
            print_usage();
            exit(0);

        default :
          printf("[Invalid Input]\n Use -h for help\n");
          break;

        case -1:
          break;
      }

      break;
    }

    if(use_log_scale) {
        args.nr = (1 << args.nr);
        args.ns = (1 << args.ns);
        args.unique_keys = (1 << args.unique_keys);
    }

    args.check();
    args.print();
}

template<typename TupleR, typename TupleS, typename TupleOut>
void free_tuple_mem(TupleR r, TupleS s, TupleOut out){
    r.free_mem();
    s.free_mem();
    out.free_mem();
}


void prepare_running(int argc, char** argv) {
#ifndef COL_T_8B
    using col_t = int;
#else
    using col_t = long;
#endif

#ifndef KEY_T_8B
    using join_key_t = int;
#else
    using join_key_t = long;
#endif

    DECL_TUP_1_TO_8(join_key_t, col_t)

    struct join_args args;
    parse_args(argc, argv, args);
}

void say_hello(){
    cout << "hello \n";
}